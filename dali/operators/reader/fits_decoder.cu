#include "hip/hip_runtime.h"
#include <iostream>
#include "dali/operators/reader/fits_decoder.cuh"

namespace dali {

__global__ void rice_decompress(unsigned char **compressed_data, void *uncompressed_data,
                                int bytepix, int blocksize, long tiles, long maxtilelen,
                                const int *tile_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  const int nonzero_count[256] = {
      0, 1, 2, 2, 3, 3, 3, 3, 4, 4, 4, 4, 4, 4, 4, 4, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5,
      5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6,
      6, 6, 6, 6, 6, 6, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7,
      7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7,
      7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8,
      8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8,
      8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8,
      8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8,
      8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8, 8};

  if (bytepix == 1) {
    for (long tile = index; tile < tiles; tile += stride) {
      int i, imax;
      int k;
      int nbits, nzero, fs;
      unsigned int b, diff, lastpix;
      int fsmax, fsbits, bbits;
      int shift;
      long beg;

      fsbits = 3;
      fsmax = 6;

      bbits = 1 << fsbits;

      lastpix = compressed_data[tile][0];
      compressed_data[tile] += 1;
      shift = 1;

      b = *compressed_data[tile]++;
      shift++;

      beg = tile * maxtilelen;
      nbits = 8;
      for (i = 0; i < tile_size[tile];) {
        nbits -= fsbits;
        while (nbits < 0) {
          b = (b << 8) | (*compressed_data[tile]++);
          shift++;
          nbits += 8;
        }
        fs = (b >> nbits) - 1;

        b &= (1 << nbits) - 1;
        imax = i + blocksize;
        if (imax > tile_size[tile])
          imax = tile_size[tile];
        if (fs < 0) {
          for (; i < imax; i++)
            ((unsigned char *)uncompressed_data)[beg + i] = lastpix;
        } else if (fs == fsmax) {
          for (; i < imax; i++) {
            k = bbits - nbits;
            diff = b << k;
            for (k -= 8; k >= 0; k -= 8) {
              b = *compressed_data[tile]++;
              shift++;
              diff |= b << k;
            }
            if (nbits > 0) {
              b = *compressed_data[tile]++;
              shift++;
              diff |= b >> (-k);
              b &= (1 << nbits) - 1;
            } else {
              b = 0;
            }

            if ((diff & 1) == 0) {
              diff = diff >> 1;
            } else {
              diff = ~(diff >> 1);
            }
            ((unsigned char *)uncompressed_data)[beg + i] = diff + lastpix;
            lastpix = ((unsigned char *)uncompressed_data)[beg + i];
          }
        } else {
          for (; i < imax; i++) {
            while (b == 0) {
              nbits += 8;
              b = *compressed_data[tile]++;
              shift++;
            }
            nzero = nbits - nonzero_count[b];
            nbits -= nzero + 1;
            b ^= 1 << nbits;
            nbits -= fs;
            while (nbits < 0) {
              b = (b << 8) | (*compressed_data[tile]++);
              shift++;
              nbits += 8;
            }
            diff = (nzero << fs) | (b >> nbits);
            b &= (1 << nbits) - 1;

            if ((diff & 1) == 0) {
              diff = diff >> 1;
            } else {
              diff = ~(diff >> 1);
            }
            ((unsigned char *)uncompressed_data)[beg + i] = diff + lastpix;
            lastpix = ((unsigned char *)uncompressed_data)[beg + i];
          }
        }
      }
      compressed_data[tile] -= shift;
    }
  } else if (bytepix == 2) {
    for (long tile = index; tile < tiles; tile += stride) {
      int i, imax, k;
      int nbits, nzero, fs;
      unsigned char bytevalue;
      unsigned int b, diff, lastpix;
      int fsmax, fsbits, bbits;
      int shift;
      long beg;

      fsbits = 4;
      fsmax = 14;

      bbits = 1 << fsbits;

      lastpix = 0;
      bytevalue = compressed_data[tile][0];
      lastpix = lastpix | (bytevalue << 8);
      bytevalue = compressed_data[tile][1];
      lastpix = lastpix | bytevalue;

      compressed_data[tile] += 2;
      shift = 2;

      b = *compressed_data[tile]++;
      shift++;

      beg = tile * maxtilelen;
      nbits = 8;
      for (i = 0; i < tile_size[tile];) {
        nbits -= fsbits;
        while (nbits < 0) {
          b = (b << 8) | (*compressed_data[tile]++);
          shift++;
          nbits += 8;
        }
        fs = (b >> nbits) - 1;

        b &= (1 << nbits) - 1;
        imax = i + blocksize;
        if (imax > tile_size[tile])
          imax = tile_size[tile];
        if (fs < 0) {
          for (; i < imax; i++)
            ((unsigned short *)uncompressed_data)[beg + i] = lastpix;
        } else if (fs == fsmax) {
          for (; i < imax; i++) {
            k = bbits - nbits;
            diff = b << k;
            for (k -= 8; k >= 0; k -= 8) {
              b = *compressed_data[tile]++;
              shift++;
              diff |= b << k;
            }
            if (nbits > 0) {
              b = *compressed_data[tile]++;
              shift++;
              diff |= b >> (-k);
              b &= (1 << nbits) - 1;
            } else {
              b = 0;
            }

            if ((diff & 1) == 0) {
              diff = diff >> 1;
            } else {
              diff = ~(diff >> 1);
            }
            ((unsigned short *)uncompressed_data)[beg + i] = diff + lastpix;
            lastpix = ((unsigned short *)uncompressed_data)[beg + i];
          }
        } else {
          for (; i < imax; i++) {
            while (b == 0) {
              nbits += 8;
              b = *compressed_data[tile]++;
              shift++;
            }
            nzero = nbits - nonzero_count[b];
            nbits -= nzero + 1;
            b ^= 1 << nbits;
            nbits -= fs;
            while (nbits < 0) {
              b = (b << 8) | (*compressed_data[tile]++);
              shift++;
              nbits += 8;
            }
            diff = (nzero << fs) | (b >> nbits);
            b &= (1 << nbits) - 1;

            if ((diff & 1) == 0) {
              diff = diff >> 1;
            } else {
              diff = ~(diff >> 1);
            }
            ((unsigned short *)uncompressed_data)[beg + i] = diff + lastpix;
            lastpix = ((unsigned short *)uncompressed_data)[beg + i];
          }
        }
      }
      compressed_data[tile] -= shift;
    }
  } else {
    for (long tile = index; tile < tiles; tile += stride) {
      int i, imax, k;
      int nbits, nzero, fs;
      unsigned char bytevalue;
      unsigned int b, diff, lastpix;
      int fsmax, fsbits, bbits;
      int shift;
      long beg;

      fsbits = 5;
      fsmax = 25;

      bbits = 1 << fsbits;

      lastpix = 0;
      bytevalue = compressed_data[tile][0];
      lastpix = lastpix | (bytevalue << 24);
      bytevalue = compressed_data[tile][1];
      lastpix = lastpix | (bytevalue << 16);
      bytevalue = compressed_data[tile][2];
      lastpix = lastpix | (bytevalue << 8);
      bytevalue = compressed_data[tile][3];
      lastpix = lastpix | bytevalue;

      compressed_data[tile] += 4;
      shift = 4;

      b = *compressed_data[tile]++;
      shift++;

      beg = tile * maxtilelen;
      nbits = 8;
      for (i = 0; i < tile_size[tile];) {
        nbits -= fsbits;
        while (nbits < 0) {
          b = (b << 8) | (*compressed_data[tile]++);
          shift++;
          nbits += 8;
        }
        fs = (b >> nbits) - 1;

        b &= (1 << nbits) - 1;
        imax = i + blocksize;
        if (imax > tile_size[tile])
          imax = tile_size[tile];
        if (fs < 0) {
          for (; i < imax; i++)
            ((unsigned int *)uncompressed_data)[beg + i] = lastpix;
        } else if (fs == fsmax) {
          for (; i < imax; i++) {
            k = bbits - nbits;
            diff = b << k;
            for (k -= 8; k >= 0; k -= 8) {
              b = *compressed_data[tile]++;
              shift++;
              diff |= b << k;
            }
            if (nbits > 0) {
              b = *compressed_data[tile]++;
              shift++;
              diff |= b >> (-k);
              b &= (1 << nbits) - 1;
            } else {
              b = 0;
            }

            if ((diff & 1) == 0) {
              diff = diff >> 1;
            } else {
              diff = ~(diff >> 1);
            }
            ((unsigned int *)uncompressed_data)[beg + i] = diff + lastpix;
            lastpix = ((unsigned int *)uncompressed_data)[beg + i];
          }
        } else {
          for (; i < imax; i++) {
            while (b == 0) {
              nbits += 8;
              b = *compressed_data[tile]++;
              shift++;
            }
            nzero = nbits - nonzero_count[b];
            nbits -= nzero + 1;
            b ^= 1 << nbits;
            nbits -= fs;
            while (nbits < 0) {
              b = (b << 8) | (*compressed_data[tile]++);
              shift++;
              nbits += 8;
            }
            diff = (nzero << fs) | (b >> nbits);
            b &= (1 << nbits) - 1;

            if ((diff & 1) == 0) {
              diff = diff >> 1;
            } else {
              diff = ~(diff >> 1);
            }
            ((unsigned int *)uncompressed_data)[beg + i] = diff + lastpix;
            lastpix = ((unsigned int *)uncompressed_data)[beg + i];
          }
        }
      }
      compressed_data[tile] -= shift;
    }
  }
}

}  // namespace dali
